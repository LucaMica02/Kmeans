/*
 * k-Means clustering algorithm
 *
 * CUDA version
 *
 * Parallel computing (Degree in Computer Engineering)
 * 2022/2023
 *
 * Version: 1.0
 *
 * (c) 2022 Diego García-Álvarez, Arturo Gonzalez-Escribano
 * Grupo Trasgo, Universidad de Valladolid (Spain)
 *
 * This work is licensed under a Creative Commons Attribution-ShareAlike 4.0 International License.
 * https://creativecommons.org/licenses/by-sa/4.0/
 */
#include <stdio.h>
#include <stdlib.h>
#include <ctype.h>
#include <math.h>
#include <time.h>
#include <string.h>
#include <float.h>
#include <hip/hip_runtime.h>

#define MAXLINE 2000
#define MAXCAD 200

// Macros
#define MIN(a, b) ((a) < (b) ? (a) : (b))
#define MAX(a, b) ((a) > (b) ? (a) : (b))

/*
 * Macros to show errors when calling a CUDA library function,
 * or after launching a kernel
 */
#define CHECK_CUDA_CALL(a)                                                                            \
	{                                                                                                 \
		hipError_t ok = a;                                                                           \
		if (ok != hipSuccess)                                                                        \
			fprintf(stderr, "-- Error CUDA call in line %d: %s\n", __LINE__, hipGetErrorString(ok)); \
	}
#define CHECK_CUDA_LAST()                                                                             \
	{                                                                                                 \
		hipError_t ok = hipGetLastError();                                                          \
		if (ok != hipSuccess)                                                                        \
			fprintf(stderr, "-- Error CUDA last in line %d: %s\n", __LINE__, hipGetErrorString(ok)); \
	}

/*
Function showFileError: It displays the corresponding error during file reading.
*/
void showFileError(int error, char *filename)
{
	printf("Error\n");
	switch (error)
	{
	case -1:
		fprintf(stderr, "\tFile %s has too many columns.\n", filename);
		fprintf(stderr, "\tThe maximum number of columns has been exceeded. MAXLINE: %d.\n", MAXLINE);
		break;
	case -2:
		fprintf(stderr, "Error reading file: %s.\n", filename);
		break;
	case -3:
		fprintf(stderr, "Error writing file: %s.\n", filename);
		break;
	}
	fflush(stderr);
}

/*
Function readInput: It reads the file to determine the number of rows and columns.
*/
int readInput(char *filename, int *lines, int *samples)
{
	FILE *fp;
	char line[MAXLINE] = "";
	char *ptr;
	const char *delim = "\t";
	int contlines, contsamples = 0;

	contlines = 0;

	if ((fp = fopen(filename, "r")) != NULL)
	{
		while (fgets(line, MAXLINE, fp) != NULL)
		{
			if (strchr(line, '\n') == NULL)
			{
				return -1;
			}
			contlines++;
			ptr = strtok(line, delim);
			contsamples = 0;
			while (ptr != NULL)
			{
				contsamples++;
				ptr = strtok(NULL, delim);
			}
		}
		fclose(fp);
		*lines = contlines;
		*samples = contsamples;
		return 0;
	}
	else
	{
		return -2;
	}
}

/*
Function readInput2: It loads data from file.
*/
int readInput2(char *filename, float *data)
{
	FILE *fp;
	char line[MAXLINE] = "";
	char *ptr;
	const char *delim = "\t";
	int i = 0;

	if ((fp = fopen(filename, "rt")) != NULL)
	{
		while (fgets(line, MAXLINE, fp) != NULL)
		{
			ptr = strtok(line, delim);
			while (ptr != NULL)
			{
				data[i] = atof(ptr);
				i++;
				ptr = strtok(NULL, delim);
			}
		}
		fclose(fp);
		return 0;
	}
	else
	{
		return -2; // No file found
	}
}

/*
Function writeResult: It writes in the output file the cluster of each sample (point).
*/
int writeResult(int *classMap, int lines, const char *filename)
{
	FILE *fp;

	if ((fp = fopen(filename, "wt")) != NULL)
	{
		for (int i = 0; i < lines; i++)
		{
			fprintf(fp, "%d\n", classMap[i]);
		}
		fclose(fp);

		return 0;
	}
	else
	{
		return -3; // No file found
	}
}

/*

Function initCentroids: This function copies the values of the initial centroids, using their
position in the input data structure as a reference map.
*/
void initCentroids(const float *data, float *centroids, int *centroidPos, int samples, int K)
{
	int i;
	int idx;
	for (i = 0; i < K; i++)
	{
		idx = centroidPos[i];
		memcpy(&centroids[i * samples], &data[idx * samples], (samples * sizeof(float)));
	}
}

/*
Function euclideanDistance: Euclidean distance
This function could be modified
*/
__device__ float euclideanDistance(float *point, float *center, int samples)
{
	float dist = 0.0;
	for (int i = 0; i < samples; i++)
	{
		dist += (point[i] - center[i]) * (point[i] - center[i]);
	}
	dist = sqrt(dist);
	return (dist);
}

/*
Function zeroFloatMatriz: Set matrix elements to 0
This function could be modified
*/
void zeroFloatMatriz(float *matrix, int rows, int columns)
{
	int i, j;
	for (i = 0; i < rows; i++)
		for (j = 0; j < columns; j++)
			matrix[i * columns + j] = 0.0;
}

/*
Function zeroIntArray: Set array elements to 0
This function could be modified
*/
void zeroIntArray(int *array, int size)
{
	int i;
	for (i = 0; i < size; i++)
		array[i] = 0;
}

// KERNEL FUNCTION
__global__ void assignPointsToCentroids(float *data, float *centroids, int *classMap, float *auxCentroids, int *pointsPerClass, int lines, int samples, int K, int *changes)
{
	extern __shared__ float sharedMemory[]; // Shared memory declaration

	// Pointers for shared memory
	float *sharedCentroids = sharedMemory;

	int tid = threadIdx.x;
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	// Load centroids into shared memory
	for (int i = tid; i < K * samples; i += blockDim.x)
	{
		sharedCentroids[i] = centroids[i];
	}
	__syncthreads(); // Ensure all threads have loaded centroids

	if (idx < lines)
	{

		// Calculate the closest centroid
		int cluster = 1;
		float minDist = FLT_MAX;

		for (int j = 0; j < K; j++)
		{
			float dist = euclideanDistance(&data[idx * samples], &sharedCentroids[j * samples], samples);
			if (dist < minDist)
			{
				minDist = dist;
				cluster = j + 1;
			}
		}

		if (classMap[idx] != cluster)
		{
			atomicAdd(changes, 1);
		}

		classMap[idx] = cluster;
		cluster--;
		atomicAdd(&pointsPerClass[cluster], 1);
		for (int j = 0; j < samples; j++)
		{
			atomicAdd(&auxCentroids[cluster * samples + j], data[idx * samples + j]);
		}
	}
}

__global__ void normalizeCentroids(float *centroids, float *auxCentroids, float *distCentroids, int *pointsPerClass, int samples, int K)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < K * samples)
	{
		int cluster = idx / samples;
		auxCentroids[idx] /= pointsPerClass[cluster];
	}

	if (idx < K)
	{
		distCentroids[idx] = euclideanDistance(&centroids[idx * samples], &auxCentroids[idx * samples], samples);
	}
}

int main(int argc, char *argv[])
{

	// START CLOCK***************************************
	clock_t start, end;
	start = clock();
	//**************************************************
	/*
	 * PARAMETERS
	 *
	 * argv[1]: Input data file
	 * argv[2]: Number of clusters
	 * argv[3]: Maximum number of iterations of the method. Algorithm termination condition.
	 * argv[4]: Minimum percentage of class changes. Algorithm termination condition.
	 *          If between one iteration and the next, the percentage of class changes is less than
	 *          this percentage, the algorithm stops.
	 * argv[5]: Precision in the centroid distance after the update.
	 *          It is an algorithm termination condition. If between one iteration of the algorithm
	 *          and the next, the maximum distance between centroids is less than this precision, the
	 *          algorithm stops.
	 * argv[6]: Output file. Class assigned to each point of the input file.
	 * */
	if (argc != 7)
	{
		fprintf(stderr, "EXECUTION ERROR K-MEANS: Parameters are not correct.\n");
		fprintf(stderr, "./KMEANS [Input Filename] [Number of clusters] [Number of iterations] [Number of changes] [Threshold] [Output data file]\n");
		fflush(stderr);
		exit(-1);
	}

	// Reading the input data
	// lines = number of points; samples = number of dimensions per point
	int lines = 0, samples = 0;

	int error = readInput(argv[1], &lines, &samples);
	if (error != 0)
	{
		showFileError(error, argv[1]);
		exit(error);
	}

	float *data = (float *)calloc(lines * samples, sizeof(float));
	if (data == NULL)
	{
		fprintf(stderr, "Memory allocation error.\n");
		exit(-4);
	}
	error = readInput2(argv[1], data);
	if (error != 0)
	{
		showFileError(error, argv[1]);
		exit(error);
	}

	// Parameters
	int K = atoi(argv[2]);
	int maxIterations = atoi(argv[3]);
	int minChanges = (int)(lines * atof(argv[4]) / 100.0);
	float maxThreshold = atof(argv[5]);

	int *centroidPos = (int *)calloc(K, sizeof(int));
	float *centroids = (float *)calloc(K * samples, sizeof(float));
	int *classMap = (int *)calloc(lines, sizeof(int));

	if (centroidPos == NULL || centroids == NULL || classMap == NULL)
	{
		fprintf(stderr, "Memory allocation error.\n");
		exit(-4);
	}

	// Initial centrodis
	srand(0);
	int i;
	for (i = 0; i < K; i++)
		centroidPos[i] = rand() % lines;

	// Loading the array of initial centroids with the data from the array data
	// The centroids are points stored in the data array.
	initCentroids(data, centroids, centroidPos, samples, K);

	printf("\n\tData file: %s \n\tPoints: %d\n\tDimensions: %d\n", argv[1], lines, samples);
	printf("\tNumber of clusters: %d\n", K);
	printf("\tMaximum number of iterations: %d\n", maxIterations);
	printf("\tMinimum number of changes: %d [%g%% of %d points]\n", minChanges, atof(argv[4]), lines);
	printf("\tMaximum centroid precision: %f\n", maxThreshold);

	// END CLOCK*****************************************
	end = clock();
	double elapsed = (double)(end - start) / CLOCKS_PER_SEC;
	printf("\nMemory allocation: %f seconds\n", elapsed);
	fflush(stdout);

	CHECK_CUDA_CALL(hipSetDevice(0));
	CHECK_CUDA_CALL(hipDeviceSynchronize());
	//**************************************************
	// START CLOCK***************************************
	start = clock();
	//**************************************************
	char *outputMsg = (char *)calloc(10000, sizeof(char));
	char line[100];

	int it = 0;
	int changes = 0;
	float maxDist;

	// pointPerClass: number of points classified in each class
	// auxCentroids: mean of the points in each class
	int *pointsPerClass = (int *)malloc(K * sizeof(int));
	float *auxCentroids = (float *)malloc(K * samples * sizeof(float));
	float *distCentroids = (float *)malloc(K * sizeof(float));
	if (pointsPerClass == NULL || auxCentroids == NULL || distCentroids == NULL)
	{
		fprintf(stderr, "Memory allocation error.\n");
		exit(-4);
	}

	/*
	 *
	 * START HERE: DO NOT CHANGE THE CODE ABOVE THIS POINT
	 *
	 */

	int *changes_d;
	float *data_d, *centroids_d, *auxCentroids_d, *distCentroids_d;
	int *classMap_d, *pointsPerClass_d;

	// Allocate device memory
	CHECK_CUDA_CALL(hipMalloc((void **)&data_d, lines * samples * sizeof(float)));
	CHECK_CUDA_CALL(hipMalloc((void **)&centroids_d, K * samples * sizeof(float)));
	CHECK_CUDA_CALL(hipMalloc((void **)&classMap_d, lines * sizeof(int)));
	CHECK_CUDA_CALL(hipMalloc((void **)&pointsPerClass_d, K * sizeof(int)));
	CHECK_CUDA_CALL(hipMalloc((void **)&auxCentroids_d, K * samples * sizeof(float)));
	CHECK_CUDA_CALL(hipMalloc((void **)&distCentroids_d, K * sizeof(float)));
	CHECK_CUDA_CALL(hipMalloc((void **)&changes_d, sizeof(int)));

	// Copy memory from host to device
	CHECK_CUDA_CALL(hipMemcpy(data_d, data, lines * samples * sizeof(float), hipMemcpyHostToDevice));
	CHECK_CUDA_CALL(hipMemcpy(centroids_d, centroids, K * samples * sizeof(float), hipMemcpyHostToDevice));
	CHECK_CUDA_CALL(hipMemcpy(classMap_d, classMap, lines * sizeof(int), hipMemcpyHostToDevice));
	CHECK_CUDA_CALL(hipMemcpy(pointsPerClass_d, pointsPerClass, K * sizeof(int), hipMemcpyHostToDevice));
	CHECK_CUDA_CALL(hipMemcpy(auxCentroids_d, auxCentroids, K * samples * sizeof(float), hipMemcpyHostToDevice));
	CHECK_CUDA_CALL(hipMemcpy(distCentroids_d, distCentroids, K * sizeof(float), hipMemcpyHostToDevice));

	do
	{
		it++;

		// Reset changes on device
		hipMemset(changes_d, 0, sizeof(int));
		hipMemset(pointsPerClass_d, 0, K * sizeof(int));
		hipMemset(auxCentroids_d, 0, K * samples * sizeof(float));

		// 1. Calculate the distance from each point to the centroid and assign to nearest centroid
		int blockSize = 32;
		int gridSize = (lines + blockSize - 1) / blockSize;
		int sharedMemorySize = (K * samples) * sizeof(float);
		assignPointsToCentroids<<<gridSize, blockSize, sharedMemorySize>>>(data_d, centroids_d, classMap_d, auxCentroids_d, pointsPerClass_d, lines, samples, K, changes_d);

		// Copy the number of changes back to host
		hipMemcpy(&changes, changes_d, sizeof(int), hipMemcpyDeviceToHost);

		// Normalize centroids
		gridSize = (K * samples + blockSize - 1) / blockSize;
		normalizeCentroids<<<gridSize, blockSize, 2>>>(centroids_d, auxCentroids_d, distCentroids_d, pointsPerClass_d, samples, K);

		// 3. Calculate maximum distance moved by centroids
		// gridSize = (K + blockSize - 1) / blockSize;
		// computeCentroidMovement<<<gridSize, blockSize, 2>>>(centroids_d, auxCentroids_d, distCentroids_d, samples, K);

		// Copy maxDist to host
		CHECK_CUDA_CALL(hipMemcpy(distCentroids, distCentroids_d, K * sizeof(float), hipMemcpyDeviceToHost));
		maxDist = distCentroids[0];
		for (int i = 1; i < K; i++)
		{
			maxDist = MAX(maxDist, distCentroids[i]);
		}

		// Update centroids for next iteration
		CHECK_CUDA_CALL(hipMemcpy(centroids_d, auxCentroids_d, K * samples * sizeof(float), hipMemcpyDeviceToDevice));

		sprintf(line, "\n[%d] Cluster changes: %d\tMax. centroid distance: %f", it, changes, maxDist);
		outputMsg = strcat(outputMsg, line);

	} while ((changes > minChanges) && (it < maxIterations) && (maxDist > maxThreshold));

	/*
	 *
	 * STOP HERE: DO NOT CHANGE THE CODE BELOW THIS POINT
	 *
	 */
	// Output and termination conditions
	printf("%s", outputMsg);

	CHECK_CUDA_CALL(hipDeviceSynchronize());

	// END CLOCK*****************************************
	end = clock();
	elapsed = (double)(end - start) / CLOCKS_PER_SEC;
	printf("\nComputation: %f seconds", elapsed);
	fflush(stdout);
	//**************************************************
	// START CLOCK***************************************
	start = clock();
	//**************************************************

	if (changes <= minChanges)
	{
		printf("\n\nTermination condition:\nMinimum number of changes reached: %d [%d]", changes, minChanges);
	}
	else if (it >= maxIterations)
	{
		printf("\n\nTermination condition:\nMaximum number of iterations reached: %d [%d]", it, maxIterations);
	}
	else
	{
		printf("\n\nTermination condition:\nCentroid update precision reached: %g [%g]", maxDist, maxThreshold);
	}

	// Writing the classification of each point to the output file.
	error = writeResult(classMap, lines, argv[6]);
	if (error != 0)
	{
		showFileError(error, argv[6]);
		exit(error);
	}

	// Free device memory
	CHECK_CUDA_CALL(hipFree(data_d));
	CHECK_CUDA_CALL(hipFree(centroids_d));
	CHECK_CUDA_CALL(hipFree(classMap_d));
	CHECK_CUDA_CALL(hipFree(pointsPerClass_d));
	CHECK_CUDA_CALL(hipFree(auxCentroids_d));
	CHECK_CUDA_CALL(hipFree(distCentroids_d));
	CHECK_CUDA_CALL(hipFree(changes_d));

	// Free host memory
	free(data);
	free(classMap);
	free(centroidPos);
	free(centroids);
	free(distCentroids);
	free(pointsPerClass);
	free(auxCentroids);

	// END CLOCK*****************************************
	end = clock();
	elapsed = (double)(end - start) / CLOCKS_PER_SEC;
	printf("\n\nMemory deallocation: %f seconds\n", elapsed);
	fflush(stdout);
	//***************************************************/
	return 0;
}
